#include "hip/hip_runtime.h"
//main.cu
#include "global.h"
#include "merge.h"

//---------------------CUDA头文件----------------
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <cuda_device_runtime_api.h>
#include <opencv2/cudev/util/vec_traits.hpp>
//---------------------CUDA头文件----------------

using namespace std;

 

//fisheye correction CUDA kernel function
__global__ void GpuMapping(PtrStepSz<uchar3> dst,PtrStepSz<uchar3> src,int ** mapGx, int ** mapGy){
	int i;  // 列id，即x坐标
	int j;  // 行id，即y坐标
	
	i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i>=dst.cols)return;
	j = threadIdx.y + blockIdx.y * blockDim.y;
	if(j>=dst.rows)return;	
	//查表映射			  
	int tx = mapGx[j][i];
	int ty = mapGy[j][i];
	if(tx<src.rows && ty<src.cols && tx>=0 && ty>=0) 
		dst(j,i) = src(tx,ty);
	// 同步所有线程	  
	__syncthreads();
}

//YUV2RGB color conversion
__device__ uchar3 YUV2RGB(uchar Y, uchar U, uchar V){
	float Yt = Y;
	float Ut = U;
	float Vt = V;
	//float rf = Yt + 1.4*(Vt - 128);
	//float gf = Yt - 0.34*(Ut - 128) - 0.71*(Vt - 128);
	//float bf = Yt + 1.77*(Ut - 128);
	float rf = 1.164* (Yt-16) + 1.596*(Vt - 128);
	rf = (rf>0 && rf<255)?rf:rf<=0? 0:255;
	float gf = 1.164* (Yt-16) - 0.813*(Ut - 128) - 0.391*(Vt - 128);
	gf = (gf>0 && gf<255)?gf:gf<=0? 0:255; 
	float bf = 1.164* (Yt-16) + 2.018*(Ut - 128);
	bf = (bf>0 && bf<255)?bf:bf<=0? 0:255;
	uchar3 res = cv::cudev::VecTraits< uchar3 >::make((uchar)bf, (uchar)gf, (uchar)rf);
	return res;
}

//YUV2RGB color conversion CUDA kernel function
__global__ void YUV420P2RGB(PtrStepSz<uchar3> dst,PtrStepSz<uchar> src){
	int i = threadIdx.x + blockIdx.x * blockDim.x; // 列id，即x坐标
	int j = threadIdx.y + blockIdx.y * blockDim.y;  // 行id，即y坐标 
	int k = 0;
	for(int m = 0; m < 2; m++){
		//判断是否越界
		if(j >=src.rows/3 || i >= src.cols/2)break;
		k = j%2; 
		for(int n = 0; n < 2; n++){		   
			dst(2*j+n,2*i+m) =  YUV2RGB(
					src(2*j+n,2*i+m),//Y通道
					src(j/2+1080,i+k*960),//U通道
					src(j/2+1350,i+k*960)//V通道
					);
		}
	}
	// 同步所有线程
	__syncthreads();
}

//RGB2YUV color conversion CUDA kernel function
__global__ void RGB2YUV420P(PtrStepSz<uchar> dst, PtrStepSz<uchar> B, PtrStepSz<uchar>G, PtrStepSz<uchar> R){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = j%2;
	if(j >= R.rows/2 || i >= R.cols/2)return;
	for(int m = 0; m < 2; m++){
		for(int n = 0; n < 2; n++){ 
			//float Y = 0.299*R(2*j+n,2*i+m)+0.587*G(2*j+n,2*i+m)+0.114*B(2*j+n,2*i+m);
			float Y = 0.257*R(2*j+n,2*i+m)+ 0.564*G(2*j+n,2*i+m) +0.098*B(2*j+n,2*i+m) +16;
			dst(2*j+n,2*i+m) =(uchar)(Y>=0 && Y<=255)?Y:Y<0? 0:255;
		}
	}	
	//float U = -0.1687*R(2*j,2*i)-0.3313*G(2*j,2*i)+0.5*B(2*j,2*i)+128;
	//float V = 0.5*R(2*j,2*i)-0.4187*G(2*j,2*i)-0.0813*B(2*j,2*i)+128;
	float U = -0.148*R(2*j,2*i) - 0.291*G(2*j,2*i) + 0.439*B(2*j,2*i) + 128;
	float V = 0.439*R(2*j,2*i) - 0.368*G(2*j,2*i) - 0.071*B(2*j,2*i) +128;
	dst(j/2+1360,i+k*2042) =(uchar)(U>=0 && U<=255)?U:U<0? 0:255;
	dst(j/2+1700,i+k*2042) =(uchar)(V>=0 && V<=255)?V:V<0? 0:255;
	__syncthreads();	
}


//load correction mapping table from file 
void mapping_init(int num, int*** mapGx, int*** mapGy) {

	if(getCudaEnabledDeviceCount()==0){
		cerr<<"此OpenCV编译的时候没有启用CUDA模块"<<endl;
	}
	
	int i,j,k;
	int tx;
	int ty;
	int *** mapData;
	int ** mapGx_host;//内存用于存储显存中一维数组的暂用数组，之后将值赋给mapGx，完成GPU中二维数组构建
	int ** mapGy_host;
	
	//allocate host(RAM) space
	mapData = (int ***)malloc(2*sizeof(int **));
	if (NULL == mapData) ;
	for (i = 0; i<2; i++) {
		mapData[i] = (int **)malloc(951*sizeof(int *));
		if (NULL == mapData[i]) ;
		for (j = 0; j<951; j++) {
			mapData[i][j] = (int *)malloc(1681*sizeof(int));
			if(NULL == mapData[i][j]) ;
		}
	}
	
	//openfile
	char fname[256];
	sprintf(fname, "mapping_table/data%d.txt",num);
	ifstream in(fname);
	
	if (! in.is_open())
	{ cout << "Error opening file"<<endl; }
	in>>tx;
	in>>ty;
	
	//allocate device(VRAM) space, and copy X coordinates' mapping table from host to device
	hipMalloc((void**)(mapGx), 951*sizeof(int*));
	mapGx_host = (int **)malloc(951*sizeof(int *));
	for (j = 0;j<951;j++) {
		for (k = 0;k<1681;k++){
			in>>mapData[0][j][k];
		}
		int* mapGx1;//一维GPU数组的指针
		hipMalloc((void**)(&mapGx1), 1681*sizeof(int));
		hipMemcpy((void*)(mapGx1), (void*)(mapData[0][j]), 1681*sizeof(int), hipMemcpyHostToDevice);//将内存中的映射表赋给一维GPU数组
		mapGx_host[j] = mapGx1;
	}
	hipMemcpy((void*)(*mapGx), (void*)(mapGx_host), 951*sizeof(int*), hipMemcpyHostToDevice);

	//allocate device(VRAM) space, and copy Y coordinates' mapping table from host to device
	hipMalloc((void**)(mapGy), 951*sizeof(int*));
	mapGy_host = (int **)malloc(951*sizeof(int *));
	for (j = 0;j<951;j++) {
		for (k = 0;k<1681;k++){
			in>>mapData[1][j][k];
		}
		int* mapGy1;//一维GPU数组的指针
		hipMalloc((void**)(&mapGy1), 1681*sizeof(int));
		hipMemcpy((void*)(mapGy1), (void*)(mapData[1][j]), 1681*sizeof(int), hipMemcpyHostToDevice);//将内存中的映射表赋给一维GPU数组
		mapGy_host[j] = mapGy1;
	}
	hipMemcpy((void*)(*mapGy), (void*)(mapGy_host), 951*sizeof(int*), hipMemcpyHostToDevice);
	//cout<<"y done"<<endl;
	in.close();
	cout<<"map"<<num<<" loading complete"<<endl;
}

//load correction mapping table from file(top camera version)
void tmapping_init(int num, int*** mapGx, int*** mapGy) {
	if(getCudaEnabledDeviceCount()==0){
		cerr<<"此OpenCV编译的时候没有启用CUDA模块"<<endl;
	}
	
	int i,j,k;//计数器
	int tx;
	int ty;
	int *** mapData;
	int ** mapGx_host;//内存用于存储显存中一维数组的暂用数组，之后将值赋给mapGx，完成GPU中二维数组构建
	int ** mapGy_host;
	
	mapData = (int ***)malloc(2*sizeof(int **));
	if (NULL == mapData) ;
	for (i = 0; i<2; i++) {
		mapData[i] = (int **)malloc(600*sizeof(int *));
		if (NULL == mapData[i]) ;
		for (j = 0; j<600; j++) {
			mapData[i][j] = (int *)malloc(4084*sizeof(int));
			if(NULL == mapData[i][j]) ;
		}
	}
	
	ifstream in("mapping_table/data1.txt");
	
	if (! in.is_open())
	{ cout << "Error opening file"<<endl; }
	//else cout<<"loading data from data"<<num<<endl;
	in>>tx;
	in>>ty;
	
	//cout<<"start x map"<<endl;
	hipMalloc((void**)(mapGx), 600*sizeof(int*));
	mapGx_host = (int **)malloc(600*sizeof(int *));
	for (j = 0;j<600;j++) {
		for (k = 0;k<4084;k++){
			in>>mapData[0][j][k];
		}
		int* mapGx1;//一维GPU数组的指针
		hipMalloc((void**)(&mapGx1), 4084*sizeof(int));
		hipMemcpy((void*)(mapGx1), (void*)(mapData[0][j]), 4084*sizeof(int), hipMemcpyHostToDevice);//将内存中的映射表赋给一维GPU数组
		mapGx_host[j] = mapGx1;
	}
	hipMemcpy((void*)(*mapGx), (void*)(mapGx_host), 600*sizeof(int*), hipMemcpyHostToDevice);
	//cout<<"x done"<<endl;
	
	
	//cout<<"start y map"<<endl;
	hipMalloc((void**)(mapGy), 600*sizeof(int*));
	mapGy_host = (int **)malloc(600*sizeof(int *));
	for (j = 0;j<600;j++) {
		for (k = 0;k<4084;k++){
			in>>mapData[1][j][k];
		}
		int* mapGy1;//一维GPU数组的指针
		hipMalloc((void**)(&mapGy1), 4084*sizeof(int));
		hipMemcpy((void*)(mapGy1), (void*)(mapData[1][j]), 4084*sizeof(int), hipMemcpyHostToDevice);//将内存中的映射表赋给一维GPU数组
		mapGy_host[j] = mapGy1;
	}
	hipMemcpy((void*)(*mapGy), (void*)(mapGy_host), 600*sizeof(int*), hipMemcpyHostToDevice);
	//cout<<"y done"<<endl;
	in.close();
	cout<<"map"<<num<<" loading complete"<<endl;
}

//fisheye correction host function
void mapping(GpuMat& gpuMat,GpuMat& gpuMatSrc,int** mapGx,int** mapGy){
	dim3 threadsPerBlock(16, //一个block有多少列
				16); //一个block有多少行
	// 计算竖直需要多少个block
	uint block_num_vertical = gpuMat.rows/threadsPerBlock.x+1;
	// 计算水平需要多少个block
	uint block_num_horizontal = gpuMat.cols/threadsPerBlock.y+1;
	dim3 numBlocks(block_num_horizontal, // 列的方向的block数目
				   	block_num_vertical);  // 行的方向的block数目
	//call GPU mapping kernel function
	GpuMapping<<<numBlocks,threadsPerBlock>>>(gpuMat,gpuMatSrc,mapGx,mapGy);
	//synchronize all threads
	hipDeviceSynchronize();
}

//YUV2RGB conversion host function
void YUV123(GpuMat& gpuMat,GpuMat& gpuMatSrc){
	dim3 threadsPerBlock(16, //一个block有多少列
				16); //一个block有多少行
	// 计算竖直需要多少个block
	uint block_num_vertical = gpuMat.rows/threadsPerBlock.y/2+1;
	//cout<<"block_num_vertical"<<block_num_vertical<<endl;
	// 计算水平需要多少个block
	uint block_num_horizontal = gpuMat.cols/threadsPerBlock.x/2+1;
	//cout<<"block_num_horizontal"<<block_num_horizontal <<endl;
	dim3 numBlocks(block_num_horizontal, // 列的方向的block数目
				   	block_num_vertical);  // 行的方向的block数目
	//call GPU mapping kernel function
	YUV420P2RGB<<<numBlocks,threadsPerBlock>>>(gpuMat,gpuMatSrc);
	//hipDeviceSynchronize();
}

//RGB2YUV conversion host function
void YUV321(GpuMat& gpuMat,GpuMat& gpuMatSrc){
	//cout<<gpuMat.size()<<gpuMatSrc.size()<<endl;
	dim3 threadsPerBlock(16, //一个block有多少列
				16); //一个block有多少行
	// 计算竖直需要多少个block
	uint block_num_vertical = gpuMatSrc.rows/threadsPerBlock.y/2+1;
	//cout<<"block_num_vertical"<<block_num_vertical<<endl;
	// 计算水平需要多少个block
	uint block_num_horizontal = gpuMatSrc.cols/threadsPerBlock.x/2+1;
		//cout<<"block_num_horizontal"<<block_num_horizontal <<endl;
	dim3 numBlocks(block_num_horizontal, // 列的方向的block数目
				   	block_num_vertical);  // 行的方向的block数目
	GpuMat gpuMatYUV[3];
	split(gpuMatSrc,gpuMatYUV);
	//call GPU mapping kernel function
	RGB2YUV420P<<<numBlocks,threadsPerBlock>>>(gpuMat,gpuMatYUV[0],gpuMatYUV[1],gpuMatYUV[2]);
	//hipDeviceSynchronize();
}



